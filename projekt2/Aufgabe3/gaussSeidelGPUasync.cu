#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#define FEHLERSCHRANKE 0.000001
//Exponent der Verfeinerung

float functionF(float x, float y);
float* allocateSquareMatrix(int size, int initialize, int n);
float* allocateVector(int size, int initialize);
void printSquareMatrix(float **matrix, int dim);
void printSquareMatrix(float *matrix, int dim);
void printVector(float *vector, int length);
void freeSquareMatrix(float **matrix, int dim);
void calculateError(float *old_val, float *new_val, int dim, float *result);
void printVectorInBlock(float *vector, int length, int blockLength);
void checkForError(const char* msg);

//const int ITERATE_ON_BLACK = 0;
//const int ITERATE_ON_RED = 1;
//const int THREADS_PER_BLOCK = 32;
const int BLOCK_DIMENSION = 8;

__global__ void blockIterationAsyncJacobi(int dim_u, float h, float* u, int iterCount) {


    __shared__ float localBlock[BLOCK_DIMENSION + 2][BLOCK_DIMENSION + 2];

    int global_start_i = (blockIdx.x * BLOCK_DIMENSION);
    int global_start_j = (blockIdx.y * BLOCK_DIMENSION);
    int global_end_i = (blockIdx.x + 1) * BLOCK_DIMENSION;
    int global_end_j = (blockIdx.y + 1) * BLOCK_DIMENSION;

    int global_i = 0;
    int global_j = 0;
    int local_i = 0;
    int local_j = 0;

    for(int iterations = 0; iterations < iterCount; iterations++) {

        // 1. Load data from global memory into local
        // load all needed data from global memory
        // 8x8 inner with neighbor elements around
        for(int i = threadIdx.x; i < (BLOCK_DIMENSION + 2) * (BLOCK_DIMENSION + 2); i += blockDim.x) {

            local_i = i % (BLOCK_DIMENSION + 2);
            local_j = (int) ( i / (BLOCK_DIMENSION + 2));
            global_i = global_start_i + local_i - 1;
            global_j = global_start_j + local_j - 1;

            if(blockIdx.x == 1 && blockIdx.y == 1) {
                printf("ThreadIdx = %d, block.x/y = %d/%d i = %d loc_(i, j) = (%d, %d), glo_(i, j) = (%d, %d)\n", threadIdx.x, global_start_i, global_start_j, i, local_i, local_j, global_i, global_j);
            }
            if(global_i >= global_start_i && global_i < global_end_i && global_j >= global_start_j && global_j <= global_end_j) {
                #ifdef PRINT
                if(blockIdx.x == 1 && blockIdx.y == 1) {
                    printf("LOAD - ThreadIdx = %d, block.x/y = %d/%d i = %d loc_(i, j) = (%d, %d), glo_(i, j) = (%d, %d)\n", threadIdx.x, global_start_i, global_start_j, i, local_i, local_j, global_i, global_j);
                }
                //printf("ThreadIdx = %d, block.x/y = %d/%d i = %d, local_i = %d, local_j = %d, global_i = %d, global_j = %d\n", threadIdx.x, blockIdx.x, blockIdx.y, i, local_i, local_j, global_i, global_j);
                #endif
                localBlock[local_i][local_j] = u[global_i + global_j * dim_u];
            }
        }
        __syncthreads();

        #ifdef PRINT

        #endif

        // 2. do jacobi iteration on local block
        int i, j;
        
        for(int iter_flag = 0; iter_flag < 2; iter_flag++) {

            int threadID = threadIdx.x * 2 + iter_flag;

            // calc the index, a thread has to work on
            // only in inner block, so add +1 to avoid iteration on neighbour elements
            j = ((int) threadID / BLOCK_DIMENSION) + 1;
            i = (((int) threadID % BLOCK_DIMENSION) + (1 - 2 * iter_flag) * (j % 2) ) + 1;
            global_i = global_start_i + i;
            global_j = global_start_j + j;        

            if(global_i >= 0 && global_i < dim_u && global_j >= 0 && global_j < dim_u) {

                #ifdef PRINT
                //printf("ThreadID = %d|x=%d|y=%d, i = %d, j = %d\n", threadID, blockIdx.x, blockIdx.y, i, j);
                #endif

                float tempSum = 
                    // top element
                    localBlock[i][j - 1] 
                    // left element
                    + localBlock[i - 1][j] 
                    // right element
                    + localBlock[i + 1][j] 
                    // bottom element
                    + localBlock[i][j + 1]; 

                // calc new value for u
                float newU = (h * h * functionF(global_i * h, global_j * h) + tempSum) / 4.0;
                // 4. replace old value
                
                localBlock[i][j] = newU;

            }
            __syncthreads();
        }
        
        #ifdef PRINT

        #endif

        // 3. write back to global memory
        // only updated values in embedded 8x8 block
        for(int i = threadIdx.x; i < BLOCK_DIMENSION * BLOCK_DIMENSION; i += blockDim.x) {

            local_i = (i % BLOCK_DIMENSION) + 1;
            local_j = (int) ( i / BLOCK_DIMENSION) + 1;
            global_i = global_start_i + local_i - 1;
            global_j = global_start_j + local_j - 1;

            if(global_i >= 0 && global_i <= dim_u && global_j >= 0 && global_j < global_end_j) {
                #ifdef PRINT
                if(blockIdx.x == 0 && blockIdx.y == 0) {
                    //printf("ThreadIdx = %d, block.x/y = %d/%d i = %d loc_(i, j) = (%d, %d), glo_(i, j) = (%d, %d)\n", threadIdx.x, global_start_i, global_start_j, i, local_i, local_j, global_i, global_j);
                }
                #endif
                u[global_i + global_j * dim_u] = localBlock[local_i][local_j];
            }
        }
        __syncthreads();
    }
    
    
    if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        printf("\n");
        for(int i = 0; i < BLOCK_DIMENSION + 2; i++) {
            for ( int j = 0; j < BLOCK_DIMENSION + 2; j++) {
                printf(" %f ", localBlock[i][j]);
            }
            printf("\n");
        }
    }
    
    __syncthreads();
    if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        printf("\n");
        for(int i = 0; i < dim_u; i++) {
            for ( int j = 0; j < dim_u; j++) {
                printf(" %f ", u[i + j * dim_u]);
            }
            printf("\n");
        }
    }
    

    __syncthreads();
}


/**
*   n: Dimension of u matrix
*   fehlerSchranke: Abbruchbedingung für Gaus Seidel Verfahren
*   h: Verfeinerung/Gitterschrittweite
*   a: pointer to 2D-Array of Matrix a (currently not used, due 
*       to extraction of calculation pattern into algorithm)
*   u: pointer to u matrix
*/
void jaccobi(int n, float fehlerSchranke, float h, float *u)
{
    //TODO: Timeranfang
    float fehler = fehlerSchranke + 1;
    float *u_new = allocateSquareMatrix(n * n, 0, n);
#ifdef PRINT
    // print embedded vector u
    printSquareMatrix(u_new, n);
#endif

    // allocate device memory
    float *gpu_u;
    hipMalloc((void**)&gpu_u, n * n * sizeof(float));
    // copy from host to device
    hipMemcpy(gpu_u, u, n * n * sizeof(float), hipMemcpyHostToDevice);
    
    // calculate the blocks per dimension
    int blocksPerDimension = n / BLOCK_DIMENSION + (n % BLOCK_DIMENSION ? 1: 0);
    int threadsPerBlock = (int) (BLOCK_DIMENSION * BLOCK_DIMENSION) / 2 + (((BLOCK_DIMENSION * BLOCK_DIMENSION) % 2) ? 2 : 0) ;
    dim3 numBlocks(blocksPerDimension, blocksPerDimension);

    printf("Running with numBlocks: %d, %d - %d Threads / Block.\n", blocksPerDimension, blocksPerDimension, threadsPerBlock);
    // Iterate as long as we do not come below our fehlerSchranke
    int count = 0;
    int block_iter = 1;
    while (fehlerSchranke < fehler)
    {
        // black iteration
        //redBlackIteration<<<numBlocks, THREADS_PER_BLOCK>>>(n, n_emb, h, gpu_u_emb, ITERATE_ON_BLACK);
        //hipDeviceSynchronize();
        // red iteration
        //redBlackIteration<<<numBlocks, THREADS_PER_BLOCK>>>(n, n_emb, h, gpu_u_emb, ITERATE_ON_RED);
        //hipDeviceSynchronize();
        blockIterationAsyncJacobi<<<numBlocks, threadsPerBlock>>>(n, h, gpu_u, block_iter);
        
        

        if(count > 1) {
            checkForError("Some shit happend.");
            hipDeviceSynchronize();
            // move result of first iteration onto host (implicitly synchronizing)
            hipMemcpy(u_new, gpu_u, n * n * sizeof(float), hipMemcpyDeviceToHost);
            
            // calculate error
            calculateError(u, u_new, n, &fehler);
            // switch pointers
            float *temp = u;
            u = u_new;
            u_new = temp;
        }

        
        // count for iterations
        count += block_iter;

        #ifdef PRINT
        printf("Iteration-Error = %.8f\n", fehler);
        printSquareMatrix(u_new, n);
        if(count > 1) {
            break;
        }
        #endif
    }
    printf("Took %d Iterations to complete.\n", count);
#ifdef PRINT
    // print embedded vector u
    printSquareMatrix(u, n);
#endif
    freeSquareMatrix(u_emb, n_emb);
    hipFree(gpu_u);
    free(u_new);
}

int main()
{

    clock_t start, stop;
    double time_used;

    start = clock();
    //Randbedingungen
    float h = 1.0;
    int n = 1;
    // calc 2^L
    for (int i = 0; i < L; i++)
    {
        n = n * 2;
    }
    h = 1.0 / (float)n;
    n = n - 1;

    printf("h = %f, n = %d, l = %d\n", h, n, L);

    //Lösungsvektoren u
    float *u = allocateVector(n * n, 1);

#ifdef PRINT
    printVector(u, (n * n));
#endif

    
    // executing gauss seidel verfahren
    jaccobi(n, FEHLERSCHRANKE, h, u);

    stop = clock();
    time_used = (double) (stop - start) / CLOCKS_PER_SEC;

    printf("Time used %f\n", time_used);

    //printVectorInBlock(u, (n * n), n);
    printVector(u, (n * n));
    free(u);

    return 0;
}

/**
 * Calculating distance between two vectors via L2-Norm  
 */
void calculateError(float* old_val, float* new_val, int dim, float *result) {

    float sum = 0.0;
    #pragma omp parallel for shared(old_val, new_val) reduction(+: sum)
    for(int i = 0; i < dim * dim; i++) {
        sum += (new_val[i] - old_val[i]) * (new_val[i] - old_val[i]);
    }
    *result = sqrtf(sum);
    //*result = temp_glob;
}

__host__ __device__
float functionF(float x, float y)
{ // x and y should be in (0,1)
    return 32.0f * (x * (1.0f - x) + y * (1.0f - y));
}

float* allocateSquareMatrix(int size, int initialize, int dim_n)
{
    float *tmp = (float*) malloc(size * sizeof(float));

    if (initialize)
    {
        for (int i = 0; i < dim_n; i++)
        {
            for (int j = 0; j < dim_n; j++)
            {
                tmp[i + j * dim_n] = 0.0;
                if (i == j)
                    tmp[i + j * dim_n] = 4.0;

                if (i + dim_n == j || i == j + dim_n || i + 1 == j || i == j + 1)
                    tmp[i + j * dim_n] = -1.0;

                if ((i % dim_n == 0 && j == i - 1) || (i == j - 1 && j % dim_n == 0))
                    tmp[i + j * dim_n] = 0.0;
            }
        }
    }
    else
    {
        for (int i = 0; i < dim_n; i++)
        {
            for (int j = 0; j < dim_n; j++)
            {
                tmp[i + j * dim_n] = 0.0;
            }
        }
    }
    return tmp;
}

/**
 *  Only frees the "rows" of the allocated Matrix. 
 *  Still have to call free on pointer of pointers
 */
void freeSquareMatrix(float **matrix, int dim)
{
    for (int i = 0; i < dim; i++)
    {
        free(matrix[i]);
    }
}

float* allocateVector(int size, int initialize)
{
    float *tmp = (float *)malloc(size * sizeof(float));
    if (initialize)
    {
        for (int i = 0; i < size; i++)
        {
            tmp[i] = 0.0;
        }
    }
    return tmp;
}
void printSquareMatrix(float **matrix, int dim)
{
    printf("Printing sqare matrix with dim = %d\n", dim);
    for (int i = 0; i < dim; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            printf(" %f", matrix[i][j]);
        }
        printf(" \n");
    }
}

void printSquareMatrix(float *matrix, int dim) {
    printf("Printing sqare matrix with dim = %d\n", dim);
    for (int i = 0; i < dim; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            printf(" %f", matrix[i + j * dim]);
        }
        printf(" \n");
    }
}

void printVector(float *vector, int length)
{
    printf("Printing Vector with length = %d\n", length);
    for (int i = 0; i < length; i++)
        printf(" %f", vector[i]);

    printf(" \n");
}

void printVectorInBlock(float *vector, int length, int blockLength)
{
    printf("Printing Vector with length = %d\n", length);
    for (int i = 0; i < length / blockLength; i++)
    {
        for (int j = 0; j < blockLength; j++)
        {
            printf(" %f", vector[i + blockLength * j]);
        }
        printf(" \n");
    }
}

void checkForError(const char* msg) {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        printf("ERROR: %s: %s\n", msg, hipGetErrorString(error));
}
