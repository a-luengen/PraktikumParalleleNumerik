#include "hip/hip_runtime.h"
#include <stdio.h>

#include <stdlib.h>

#define FEHLERSCHRANKE 0.000001
//Exponent der Verfeinerung

float functionF(float x, float y);
float* allocateSquareMatrix(int size, int initialize, int n);
float* allocateVector(int size, int initialize);
void printSquareMatrix(float **matrix, int dim);
void printSquareMatrix(float *matrix, int dim);
void printVector(float *vector, int length);
void freeSquareMatrix(float **matrix, int dim);
float calculateError(float *old_val, float *new_val, int dim);
void printVectorInBlock(float *vector, int length, int blockLength);
void checkForError(const char* msg);

const int ITERATE_ON_BLACK = 0;
const int ITERATE_ON_RED = 1;
const int THREADS_PER_BLOCK = 32;
const int BLOCK_DIMENSION = 8;

/**
*   expecting embedded result-matrix to iterate on
*   dim: dimension of the matrix u (without embedding)
*   ITERATION_FLAG: has value 0 to iterate on "black" elements or 1 to iterate on "red" elements
*/
__global__ void redBlackIteration(int dim_u, int dim_u_emb, float h, float* u_emb, int ITERATION_FLAG) {


    int threadID = threadIdx.x * 2 + ITERATION_FLAG;
    int i_offset = blockDim.x * blockIdx.x;
    int j_offset = blockDim.y * blockIdx.y;
    // use index of thread to calculate position in matrix u_emb 
    // to execute computation on
    int j_inner, i_emb, j_emb;

    // 1. Calculate the index's of embedded matrix to a thread has to work on
    j_emb = j_offset + (int) threadID / blockDim.x;
    i_emb = i_offset + (int) threadID % 8 + (1 - 2 * ITERATION_FLAG) * (j_emb % 2);
    
    printf("threadID = %d, j_emb = %d, i_emb = %d\n", threadID, j_emb, i_emb);
    if(i_emb < dim_u_emb - 1 && i_emb > 0 && j_emb < dim_u_emb - 1 && j_emb > 0) {
        // 2. calculate the index's of inner matrix for the functionF-call
        //i_inner = i_emb - 1;
        j_inner = j_emb - 1;
        printf("I calculate - threadID = %d, i_offset = %d, j_offset = %d, j_emb = %d, i_emb = %d, j_inner = %d\n", threadID, i_offset, j_offset, j_emb, i_emb, j_inner);
        // 3. calculate new value for u_emb
        float tempSum = 
            // top element
            u_emb[i_emb + (j_emb - 1) * dim_u_emb] 
            // left element
            + u_emb[i_emb - 1 + j_emb * dim_u_emb] 
            // right element
            + u_emb[i_emb + 1 + j_emb * dim_u_emb] 
            // bottom element
            + u_emb[i_emb + (j_emb + 1) * dim_u_emb]; 

        // calc new value for u
        float newU = (h * h * functionF((j_inner / dim_u + 1) * h, (j_inner % dim_u + 1) * h) + tempSum) / 4.0;
        // 4. replace old value
        u_emb[i_emb + j_emb * dim_u_emb] = newU;
    }
}

/**
*      
*   n: Dimension of u matrix
*   fehlerSchranke: Abbruchbedingung für Gaus Seidel Verfahren
*   h: Verfeinerung/Gitterschrittweite
*   a: pointer to 2D-Array of Matrix a (currently not used, due 
*       to extraction of calculation pattern into algorithm)
*   u: pointer to u matrix
*/
void gaussSeidel(int n, float fehlerSchranke, float h, float *u)
{
    //TODO: Timeranfang
    float fehler = fehlerSchranke + 1;

    // embedd vector u for corner case
    // u(0, y) = u(1, y) = u(x, 0) = u(y, 1) = 0.0
    int n_emb = n + 2;
    float *u_emb = allocateSquareMatrix(n_emb * n_emb, 0, n_emb);
    float *u_emb_new = allocateSquareMatrix(n_emb * n_emb, 0, n_emb);

    for (int i = 0; i < n_emb; i++)
    {
        for (int j = 0; j < n_emb; j++)
        {
            if (j == 0 || i == 0 || j == n_emb || i == n_emb)
            {
                // fill up with edge value
                u_emb[i + j * n_emb] = 0.0;
            }
            else
            {
                // copy value from u
                u_emb[i + j * n_emb] = u[i - 1 + n * j - 1];
            }
        }
    }
#ifdef PRINT
    // print embedded vector u
    printSquareMatrix(u_emb, n_emb);
#endif

    // allocate device memory
    float *gpu_u_emb;
    hipMalloc((void**)&gpu_u_emb, n_emb * n_emb * sizeof(float));
    // copy from local to device
    hipMemcpy(gpu_u_emb, u_emb, n_emb * n_emb * sizeof(float), hipMemcpyHostToDevice);
    checkForError("After Copying data to device.");
    
    // calculate the blocks per dimension
    int blocksPerDimension = 1 + n_emb / BLOCK_DIMENSION;
    dim3 numBlocks(blocksPerDimension, blocksPerDimension);

    printf("Running with numBlocks: %d, %d\n and %d of Threads per Block.\n", blocksPerDimension, blocksPerDimension, THREADS_PER_BLOCK);
    // Iterate as long as we do not come below our fehelrSchranke
    while (fehlerSchranke < fehler)
    {
        //int dim_u, int dim_u_emb, float h, float* u_emb, int ITERATION_FLAG
        // black iteration
        redBlackIteration<<<numBlocks, THREADS_PER_BLOCK>>>(n, n_emb, h, u_emb, ITERATE_ON_BLACK);
        // red iteration
        redBlackIteration<<<numBlocks, THREADS_PER_BLOCK>>>(n, n_emb, h, u_emb, ITERATE_ON_RED);
        hipDeviceSynchronize();
        checkForError("After 2 Kernel Executions");
        // move result of first iteration onto host
        hipMemcpy(u_emb_new, gpu_u_emb, n_emb * n_emb * sizeof(float), hipMemcpyDeviceToHost);
        
        // calculate error
        fehler = calculateError(u_emb, u_emb_new, n_emb);
        // switch pointers
        float temp = *u_emb;
        *u_emb = *u_emb_new;
        *u_emb_new = temp;

        #ifdef PRINT
        printf("Iteration-Error = %0.0f\n", fehler);
        printSquareMatrix(u_emb, n_emb);
        #endif
    }

#ifdef PRINT
    // print embedded vector u
    printSquareMatrix(u_emb, n_emb);
#endif

    // get values out of embedded vector
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            u[i + j * n] = u_emb[i + 1 + (j + 1) * n_emb];
        }
    }
    //freeSquareMatrix(u_emb, n_emb);
    free(u_emb);
}

int main()
{
    //Randbedingungen
    float h = 1.0;
    int n = 1;
    // calc 2^L
    for (int i = 0; i < L; i++)
    {
        n = n * 2;
    }
    h = 1.0 / (float)n;
    n = n - 1;

    printf("h = %f, n = %d, l = %d\n", h, n, L);

    //Lösungsvektoren u
    float *u = allocateVector(n * n, 1);

#ifdef PRINT
    //printSquareMatrix(a, (n * n)*(n * n));
    printVector(u, (n * n));
#endif

    // executing gauss seidel verfahren
    gaussSeidel(n, FEHLERSCHRANKE, h, u);

    printVectorInBlock(u, (n * n), n);
    printVector(u, (n * n));
    free(u);
    return 0;
}

float calculateError(float* old_val, float* new_val, int dim) {
    float temp_glob = 0.0;
    float temp_loc = 0.0;
    for(int i = 0; i < dim * dim; i++) {
        temp_loc = old_val[i] - new_val[i];
        if(temp_loc < 0)
            temp_loc = -temp_loc;
        if(temp_loc > temp_glob)
            temp_glob = temp_loc;
    }
    return temp_glob;
}

__host__ __device__
float functionF(float x, float y)
{ // x and y should be in (0,1)
    return 32.0f * (x * (1.0f - x) + y * (1.0f - y));
}

float* allocateSquareMatrix(int size, int initialize, int dim_n)
{
    float *tmp = (float*) malloc(size * sizeof(float));

    if (initialize)
    {
        for (int i = 0; i < dim_n; i++)
        {
            for (int j = 0; j < dim_n; j++)
            {
                tmp[i + j * dim_n] = 0.0;
                if (i == j)
                    tmp[i + j * dim_n] = 4.0;

                if (i + dim_n == j || i == j + dim_n || i + 1 == j || i == j + 1)
                    tmp[i + j * dim_n] = -1.0;

                if ((i % dim_n == 0 && j == i - 1) || (i == j - 1 && j % dim_n == 0))
                    tmp[i + j * dim_n] = 0.0;
            }
        }
    }
    else
    {
        for (int i = 0; i < dim_n; i++)
        {
            for (int j = 0; j < dim_n; j++)
            {
                tmp[i + j * dim_n] = 0.0;
            }
        }
    }
    return tmp;
}

/**
 *  Only frees the "rows" of the allocated Matrix. 
 *  Still have to call free on pointer of pointers
 */
void freeSquareMatrix(float **matrix, int dim)
{
    for (int i = 0; i < dim; i++)
    {
        free(matrix[i]);
    }
}

float* allocateVector(int size, int initialize)
{
    float *tmp = (float *)malloc(size * sizeof(float));
    if (initialize)
    {
        for (int i = 0; i < size; i++)
        {
            tmp[i] = 0.0;
        }
    }
    return tmp;
}
void printSquareMatrix(float **matrix, int dim)
{
    printf("Printing sqare matrix with dim = %d\n", dim);
    for (int i = 0; i < dim; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            printf("|%f", matrix[i][j]);
        }
        printf("|\n");
    }
}

void printSquareMatrix(float *matrix, int dim) {
    printf("Printing sqare matrix with dim = %d\n", dim);
    for (int i = 0; i < dim; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            printf("|%f", matrix[i + j * dim]);
        }
        printf("|\n");
    }
}

void printVector(float *vector, int length)
{
    printf("Printing Vector with length = %d\n", length);
    for (int i = 0; i < length; i++)
        printf("|%f", vector[i]);

    printf("|\n");
}

void printVectorInBlock(float *vector, int length, int blockLength)
{
    printf("Printing Vector with length = %d\n", length);
    for (int i = 0; i < length / blockLength; i++)
    {
        for (int j = 0; j < blockLength; j++)
        {
            printf("|%f", vector[i + blockLength * j]);
        }
        printf("|\n");
    }
}

void checkForError(const char* msg) {
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
        printf("ERROR: %s: %s\n", msg, hipGetErrorString(error));
}