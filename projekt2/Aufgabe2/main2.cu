#include "hip/hip_runtime.h"
#include <stdio.h>

#define ARRAY_SIZE 100000

__global__
void constInc(int increment, int *array, int arrayLength) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < n) array[i] = array[i] + increment;

}

/**
*  Print an Array with certain length on console
*/
void printArray(int *array, int length) {
        return;
        for(int i = 0; i < length; i++) {
                printf("|%d", array[i]);
        }
        printf("|\n");
}

/**
* Increment each element of the given Array with a certain constant on the GPU
*/
void incrementOnGPU(int *array, int arLength, int constant) {
	int blockSize = 32;
	int blocks = 32;
	
	int* gpu_arr;
	hipMalloc(&gpu_arr, arLength * sizeof(int));

}

int main(void) {

        hipSetDevice(0);

        int dataBits = ARRAY_SIZE * sizeof(int) * 8;

        printf("Copy Between GPU and Host System Test. \n");
        printf("Array Size : %d Entries\n", ARRAY_SIZE);
        printf("Memory Size: %d bytes\n", dataBits / 8);


        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float ms = 0.0f;


        int *host1 = (int*) malloc(sizeof(int) * ARRAY_SIZE);


        // init host1 array
        for(int i = 0; i < ARRAY_SIZE; i++) {
                host1[i] = i + 1;
        }

        printArray(host1, ARRAY_SIZE);

        // copy on host
        hipEventRecord(start);

        memcpy(host2, host1, sizeof(int) * ARRAY_SIZE);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);

        //printArray(host2, ARRAY_SIZE);
        printf("Host1 to Host2 took: %.8fms - %013.2f Bit/s \n", ms, ( dataBits / ms ));


        // copy from host to gpu
        hipEventRecord(start);

        hipMemcpy(gpu1, host2, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        printf("Host2 to GPU1  took: %.8fms - %013.2f Bit/s \n", ms, ( dataBits / ms));


        // execute increment Kernel
        printf("Executing Kernel");
        constInc<<< 32, 32>>>(5, gpu1, ARRAY_SIZE);

        // copy on gpu
        hipEventRecord(start);

        hipMemcpy(gpu2, gpu1, sizeof(int) * ARRAY_SIZE, hipMemcpyDeviceToDevice);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        printf("GPU1  to GPU2  took: %.8fms - %013.2f Bit/s \n", ms, (dataBits / ms));


        // copy from gpu to host
        hipEventRecord(start);

        hipMemcpy(host1, gpu2, sizeof(int) * ARRAY_SIZE, hipMemcpyDeviceToHost);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        printf("GPU2  to Host1 took: %.8fms - %013.2f Bit/s \n", ms, (dataBits / ms));

        printArray(host1, ARRAY_SIZE);

        free(host1);

        return 0;
}

        hipMemcpy(host1, gpu2, sizeof(int) * ARRAY_SIZE, hipMemcpyDeviceToHost);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        printf("GPU2  to Host1 took: %.8fms - %013.2f Bit/s \n", ms, (dataBits / ms));

        printArray(host1, ARRAY_SIZE);
        return 0;
}
